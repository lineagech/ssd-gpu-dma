#include <hip/hip_runtime.h>
#include "settings.h"
#include "ctrl.h"
#include "buffer.h"
#include "queue.h"
#include <nvm_util.h>
#include <nvm_error.h>
#include <nvm_types.h>
#include <nvm_admin.h>
#include <stdexcept>
#include <string>
#include <cstdint>

// CHIA-HAO: include debug.h
#include "debug.h"

using error = std::runtime_error;
using std::string;

__host__ DmaPtr prepareQueuePairOnHost(QueuePair& qp, const Controller& ctrl, const Settings& settings)
{
    size_t queueMemSize = ctrl.info.page_size * 2;
    size_t prpListSize = ctrl.info.page_size * settings.numThreads * (settings.doubleBuffered + 1);

    // qmem->vaddr will be already a device pointer after the following call
    // nvm_dma_t *qmem
    auto qmem = createDmaOnHost(ctrl.ctrl, NVM_PAGE_ALIGN(queueMemSize + prpListSize, 1UL << 16), settings.cudaDevice, settings.adapter, settings.segmentId);

    // Set members
    qp.pageSize = ctrl.info.page_size;
    qp.blockSize = ctrl.ns.lba_data_size;
    qp.nvmNamespace = ctrl.ns.ns_id;
    qp.pagesPerChunk = settings.numPages;
    qp.doubleBuffered = settings.doubleBuffered;
    
    qp.prpList = NVM_DMA_OFFSET(qmem, 2);
    qp.prpListIoAddr = qmem->ioaddrs[2];
    
    // Create completion queue
    int status = nvm_admin_cq_create(ctrl.aq_ref, &qp.cq, 1, qmem->vaddr, qmem->ioaddrs[0]);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create completion queue: ") + nvm_strerror(status));
    }

    // Get a valid device pointer for CQ doorbell
    void* devicePtr = nullptr;
    hipError_t err = hipHostGetDevicePointer(&devicePtr, (void*) qp.cq.db, 0);
    if (err != hipSuccess)
    {
        throw error(string("Failed to get device pointer") + hipGetErrorString(err));
    }
    // CHIA-HAO: for debug
    #if BENCH_DEBUG
    printf("Completin queue doorbell devicePtr %p -> %p\n", (void*)qp.cq.db, devicePtr);
    qp.cq.host_db = qp.cq.db;
    #endif /* END OF BENCH_DEBUG*/
    
    qp.cq.db = (volatile uint32_t*) devicePtr;

    // Create submission queue
    status = nvm_admin_sq_create(ctrl.aq_ref, &qp.sq, &qp.cq, 1, NVM_DMA_OFFSET(qmem, 1), qmem->ioaddrs[1]);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create submission queue: ") + nvm_strerror(status));
    }

    // Get a valid device pointer for SQ doorbell
    err = hipHostGetDevicePointer(&devicePtr, (void*) qp.sq.db, 0);
    if (err != hipSuccess)
    {
        throw error(string("Failed to get device pointer") + hipGetErrorString(err));
    }
    // CHIA-HAO: for debug
    #if BENCH_DEBUG
    printf("Submission queue doorbell devicePtr %p -> %p\n", (void*)qp.sq.db, devicePtr);
    qp.sq.host_db = qp.sq.db;
    #endif /* END OF BENCH_DEBUG*/

    qp.sq.db = (volatile uint32_t*) devicePtr;

    return qmem;
}


__host__ DmaPtr prepareQueuePair(QueuePair& qp, const Controller& ctrl, const Settings& settings)
{
    size_t queueMemSize = ctrl.info.page_size * 2;
    size_t prpListSize = ctrl.info.page_size * settings.numThreads * (settings.doubleBuffered + 1);

    // qmem->vaddr will be already a device pointer after the following call
    auto qmem = createDma(ctrl.ctrl, NVM_PAGE_ALIGN(queueMemSize + prpListSize, 1UL << 16), settings.cudaDevice, settings.adapter, settings.segmentId);
    
    // CHIA-HAO:
    fprintf(stderr, "Allocate queue mem %zu bytes\n", NVM_PAGE_ALIGN(queueMemSize + prpListSize, 1UL << 16));
    for (uint32_t i = 0; i < qmem->n_ioaddrs; i++) {
        fprintf(stderr, "queue mem: %u-th page vaddr %lx and ioaddr %lx\n",
                i, (uint64_t)qmem->vaddr+qmem->page_size*i, *(qmem->ioaddrs+i));    
    }

    // Set members
    qp.pageSize = ctrl.info.page_size;
    qp.blockSize = ctrl.ns.lba_data_size;
    qp.nvmNamespace = ctrl.ns.ns_id;
    qp.pagesPerChunk = settings.numPages;
    qp.doubleBuffered = settings.doubleBuffered;
    
    qp.prpList = NVM_DMA_OFFSET(qmem, 2);
    qp.prpListIoAddr = qmem->ioaddrs[2];
    
    // CHIA-HAO
    fprintf(stderr, "%s: queu pair prpList %p (ioaddr %lx)\n", __func__, qp.prpList, qp.prpListIoAddr);

    // Create completion queue
    int status = nvm_admin_cq_create(ctrl.aq_ref, &qp.cq, 1, qmem->vaddr, qmem->ioaddrs[0]);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create completion queue: ") + nvm_strerror(status));
    }
    // CHIA-HAO:
    fprintf(stderr, "%s: cq vaddr %p, ioaddr %lx\n", __func__, qmem->vaddr, qmem->ioaddrs[0]);

    // Get a valid device pointer for CQ doorbell
    void* devicePtr = nullptr;
    hipError_t err = hipHostGetDevicePointer(&devicePtr, (void*) qp.cq.db, 0);
    if (err != hipSuccess)
    {
        throw error(string("Failed to get device pointer") + hipGetErrorString(err));
    }
    // CHIA-HAO
    printf("Completin queue doorbell devicePtr %p\n", devicePtr);
    qp.cq.host_db = qp.cq.db;
    
    qp.cq.db = (volatile uint32_t*) devicePtr;

    // Create submission queue
    status = nvm_admin_sq_create(ctrl.aq_ref, &qp.sq, &qp.cq, 1, NVM_DMA_OFFSET(qmem, 1), qmem->ioaddrs[1]);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create submission queue: ") + nvm_strerror(status));
    }
    // CHIA-HAO:
    fprintf(stderr, "%s: cq vaddr %p, ioaddr %lx\n", __func__, NVM_DMA_OFFSET(qmem, 1), qmem->ioaddrs[1]);


    // Get a valid device pointer for SQ doorbell
    err = hipHostGetDevicePointer(&devicePtr, (void*) qp.sq.db, 0);
    if (err != hipSuccess)
    {
        throw error(string("Failed to get device pointer") + hipGetErrorString(err));
    }
    // CHIA-HAO
    printf("Submission queue doorbell devicePtr %p\n", devicePtr);
    qp.sq.host_db = qp.sq.db;

    qp.sq.db = (volatile uint32_t*) devicePtr;

    return qmem;
}

